#include "hip/hip_runtime.h"
#include <ctime>
#include <cstdio>
#include "../Interpreter.h"
#include "../m_global/S_Int.h"
#include "../m_global/I_AsFloatStar.h"
#include "../m_global/I_AsIntStar.h"
#include "../m_global/I_AsVoidStar.h"

#include "../m_cuda/M_Cuda.h"
#include "../m_cuda/S_Dim3.h"

#include "Ch2Module.h"


// =============================================================================
// Kernels

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)   C[i] = A[i] + B[i];
}

__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index"
           " %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,
           ix, iy, idx, A[idx]);
}

__global__ void sumMatrixOnGPU2DBlock2DGrid(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
        MatC[idx] = MatA[idx] + MatB[idx];
}

// grid 2D block 1D
__global__ void sumMatrixOnGPU1DBlock2DGrid(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
        MatC[idx] = MatA[idx] + MatB[idx];
}

// =============================================================================
// Words


// ( hostref gpuref num -- int )
class CheckResultWord : public Word
{
public:
    CheckResultWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int num = AsInt(interp->StackPop());
        float* gpuRef = AsFloatStar(interp->StackPop());
        float* hostRef = AsFloatStar(interp->StackPop());

        double epsilon = 1.0E-8;
        bool match = 1;

        for (int i = 0; i < num; i++) {
            if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
                match = 0;
                printf("Arrays do not match!\n");
                printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                       gpuRef[i], i);
                break;
            }
        }
        interp->StackPush(shared_ptr<S_Int>(new S_Int(match)));
    }
};


// ( addr num -- )
class InitDataWord : public Word
{
public:
    InitDataWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int num = AsInt(interp->StackPop());
        float* addr = AsFloatStar(interp->StackPop());

        for (int i = 0; i < num; i++) {
            addr[i] = (float)(rand() & 0xFF) / 10.0f;
        }
    }
};

// ( addr-A addr-B addr-C n -- )
class HSumArraysWord : public Word
{
public:
    HSumArraysWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int N = AsInt(interp->StackPop());
        auto C = AsFloatStar(interp->StackPop());
        auto B = AsFloatStar(interp->StackPop());
        auto A = AsFloatStar(interp->StackPop());

        for (int idx = 0; idx < N; idx++)    C[idx] = A[idx] + B[idx];
    }
};

// ( grid block addr-A addr-B addr-C n -- )
class DSumArraysWord : public Word
{
public:
    DSumArraysWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int N = AsInt(interp->StackPop());
        auto d_C = AsFloatStar(interp->StackPop());
        auto d_B = AsFloatStar(interp->StackPop());
        auto d_A = AsFloatStar(interp->StackPop());
        int block = AsInt(interp->StackPop());
        int grid = AsInt(interp->StackPop());

        sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, N);
    }
};


// ( addr size -- )
class InitialIntWord : public Word
{
public:
    InitialIntWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int size = AsInt(interp->StackPop());
        int* A = AsIntStar(interp->StackPop());
        for (int i=0; i < size; i++) {
            A[i] = i;
        }
    }
};


// ( addr-C nx ny -- )
/*
 * This example helps to visualize the relationship between thread/block IDs and
 * offsets into data. For each CUDA thread, this example displays the
 * intra-block thread ID, the inter-block block ID, the global coordinate of a
 * thread, the calculated offset into input data, and the input data at that
 * offset.
 */
class PrintIntMatrixWord : public Word
{
public:
    PrintIntMatrixWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int ny = AsInt(interp->StackPop());
        int nx = AsInt(interp->StackPop());
        int* C = AsIntStar(interp->StackPop());

        int *ic = C;
        printf("\nMatrix: (%d.%d)\n", nx, ny);

        for (int iy = 0; iy < ny; iy++) {
            for (int ix = 0; ix < nx; ix++) {
                printf("%3d", ic[ix]);
            }

            ic += nx;  // Advance to next row
            printf("\n");
        }

        printf("\n");
        return;
    }
};


// ( grid block addr-A nx ny -- )
class PrintThreadIndexWord : public Word
{
public:
    PrintThreadIndexWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int ny = AsInt(interp->StackPop());
        int nx = AsInt(interp->StackPop());
        int* A = AsIntStar(interp->StackPop());
        dim3 block = AsDim3(interp->StackPop());
        dim3 grid = AsDim3(interp->StackPop());

        printThreadIndex<<<grid, block>>>(A, nx, ny);
    }
};


// ( addr-A addr-B addr-C nx ny -- )
class HSumMatricesWord : public Word
{
public:
    HSumMatricesWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int ny = AsInt(interp->StackPop());
        int nx = AsInt(interp->StackPop());
        float* C = AsFloatStar(interp->StackPop());
        float* B = AsFloatStar(interp->StackPop());
        float* A = AsFloatStar(interp->StackPop());

        float *ia = A;
        float *ib = B;
        float *ic = C;

        for (int iy = 0; iy < ny; iy++) {
            for (int ix = 0; ix < nx; ix++) {
                ic[ix] = ia[ix] + ib[ix];
            }
            ia += nx; ib += nx; ic += nx;
        }
    }
};


// ( grid block  addr-A addr-B addr-C nx ny -- )
class DSumMatricesWord : public Word
{
public:
    DSumMatricesWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int ny = AsInt(interp->StackPop());
        int nx = AsInt(interp->StackPop());
        float* C = AsFloatStar(interp->StackPop());
        float* B = AsFloatStar(interp->StackPop());
        float* A = AsFloatStar(interp->StackPop());
        dim3 block = AsDim3(interp->StackPop());
        dim3 grid = AsDim3(interp->StackPop());

        sumMatrixOnGPU2DBlock2DGrid<<<grid, block>>>(A, B, C, nx, ny);
    }
};


// ( grid block  addr-A addr-B addr-C nx ny -- )
class DSumMatrices2DGrid1DBlockWord : public Word
{
public:
    DSumMatrices2DGrid1DBlockWord(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int ny = AsInt(interp->StackPop());
        int nx = AsInt(interp->StackPop());
        float* C = AsFloatStar(interp->StackPop());
        float* B = AsFloatStar(interp->StackPop());
        float* A = AsFloatStar(interp->StackPop());
        dim3 block = AsDim3(interp->StackPop());
        dim3 grid = AsDim3(interp->StackPop());

        sumMatrixOnGPU1DBlock2DGrid<<<grid, block>>>(A, B, C, nx, ny);
    }
};



// =============================================================================
// Ch2Module

Ch2Module::Ch2Module() : Module("ch2") {
    AddWord(shared_ptr<Word>(new CheckResultWord("CHECK-RESULT")));
    AddWord(shared_ptr<Word>(new InitDataWord("INIT-DATA")));
    AddWord(shared_ptr<Word>(new HSumArraysWord("H-SUM-ARRAYS")));
    AddWord(shared_ptr<Word>(new DSumArraysWord("D-SUM-ARRAYS")));

    AddWord(shared_ptr<Word>(new InitialIntWord("INITIAL-INT")));
    AddWord(shared_ptr<Word>(new PrintIntMatrixWord("PRINT-INT-MATRIX")));
    AddWord(shared_ptr<Word>(new PrintThreadIndexWord("PRINT-THREAD-INDEX")));

    AddWord(shared_ptr<Word>(new HSumMatricesWord("H-SUM-MATRICES")));
    AddWord(shared_ptr<Word>(new DSumMatricesWord("D-SUM-MATRICES")));
    AddWord(shared_ptr<Word>(new DSumMatrices2DGrid1DBlockWord("D-SUM-MATRICES-2DGRID-1DBLOCK")));
}
