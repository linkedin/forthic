#include "hip/hip_runtime.h"
#include <ctime>
#include <cstdio>
#include <cmath>

#include "../Interpreter.h"

#include "../m_global/S_Int.h"
#include "../m_global/S_Address.h"
#include "../m_global/I_AsArray.h"

#include "../m_cuda/M_Cuda.h"
#include "../m_cuda/S_Dim3.h"

#include "M_Gauss.h"


// =============================================================================
// Kernels

#define EPSILON 1E-6

__global__ void pivot(int num_rows, int num_cols, float *A, int pivot_row, int pivot_col) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int row = idx / num_cols;
    int col = idx % num_cols;

    // If thread isn't in matrix, return
    if (row >= num_rows || col >= num_cols)   return;

    int pivot_index = pivot_row * num_cols + pivot_col;
    float pivot_coeff = A[pivot_index];

    // If pivot coeff is 0, don't do anything
    if (fabs(pivot_coeff) < EPSILON)   return;

    // Normalize pivot row
    if (row == pivot_row) {
        A[idx] /= pivot_coeff;
    }

    // Synchronize so other threads can pick up the normalized coefficients
    __threadfence();

    float pivot_row_cur_col_coeff = A[pivot_row*num_cols + col];
    float cur_row_pivot_col_coeff = A[row*num_cols + pivot_col];

    // Eliminate pivot
    if (row == pivot_row)                              return;
    else if (fabs(cur_row_pivot_col_coeff) < EPSILON)  return;
    else    A[idx] += -cur_row_pivot_col_coeff * pivot_row_cur_col_coeff;
}


// =============================================================================
// Words


// ( floats num_rows num_cols  -- addr )
class W_GpuMatrix : public Word
{
public:
    W_GpuMatrix(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int num_cols = AsInt(interp->StackPop());
        int num_rows = AsInt(interp->StackPop());
        auto numbers = AsArray(interp->StackPop());

        int num_elements = num_rows * num_cols;
        int num_bytes = num_elements * sizeof(float);

        // Allocate memory
        void* result;
        auto res = hipMallocManaged((void**)&result, num_bytes);
        checkCudaCall(res, __FILE__, __LINE__);

        // Set values
        float* dst = (float*)result;
        for (int i=0; i < numbers.size(); i++) {
            dst[i] = AsFloat(numbers[i]);
        }

        interp->StackPush(S_Address::New(result));
    }
};

// ( num_rows num_cols  addr -- )
class W_PrintMatrix : public Word
{
public:
    W_PrintMatrix(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        float* A = AsFloatStar(interp->StackPop());
        int num_cols = AsInt(interp->StackPop());
        int num_rows = AsInt(interp->StackPop());

        interp->Run("CUDA-DEVICE-SYNCHRONIZE");

        for (int r=0; r < num_rows; r++) {
            for (int c=0; c < num_cols; c++) {
                int index = c + num_cols*r;
                printf("%6.2f ", A[index]);
            }
            printf("\n");
        }
    }
};


// ( grid block num_rows num_cols addr pivot_row pivot_col -- )
class W_Pivot : public Word
{
public:
    W_Pivot(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int pivot_col = AsInt(interp->StackPop());
        int pivot_row = AsInt(interp->StackPop());
        auto A = AsFloatStar(interp->StackPop());
        int num_cols = AsInt(interp->StackPop());
        int num_rows = AsInt(interp->StackPop());
        dim3 block = AsDim3(interp->StackPop());
        dim3 grid = AsDim3(interp->StackPop());

        pivot<<<grid, block>>>(num_rows, num_cols, A, pivot_row, pivot_col);
    }
};


// =============================================================================
// M_Gauss

M_Gauss::M_Gauss() : Module("gauss") {
    AddWord(new W_GpuMatrix("GPU-MATRIX"));
    AddWord(new W_PrintMatrix("PRINT-MATRIX"));
    AddWord(new W_Pivot("PIVOT"));
}
