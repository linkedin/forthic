#include "S_CudaDeviceProp.h"

shared_ptr<S_CudaDeviceProp> S_CudaDeviceProp::New(hipDeviceProp_t value) {
    return shared_ptr<S_CudaDeviceProp>(new S_CudaDeviceProp(value));
}

const hipDeviceProp_t& S_CudaDeviceProp::deviceProp() {
    return value;
}

string S_CudaDeviceProp::StringRep() {
    return "S_CudaDeviceProp";
}

string S_CudaDeviceProp::AsString() {
    return "S_CudaDeviceProp";
}
