#include "hip/hip_runtime.h"
#include <cstdio>
#include <sstream>

#include "../Interpreter.h"
#include "../m_global/S_Int.h"
#include "../m_global/S_Address.h"
#include "../S_String.h"

#include "S_Dim3.h"
#include "M_Cuda.h"
#include "S_CudaDeviceProp.h"


// =============================================================================
// Kernels
__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}


__global__ void checkIndex() {
    printf("blockIdx:(%d, %d, %d) threadIdx:(%d, %d, %d) blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n",
           blockIdx.x, blockIdx.y, blockIdx.z,
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockDim.x, blockDim.y, blockDim.z,
           gridDim.x, gridDim.y, gridDim.z);
}


// =============================================================================
// Words

// ( x y z -- dim3 )
class W_Dim3 : public Word
{
public:
    W_Dim3(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int z = AsInt(interp->StackPop());
        int y = AsInt(interp->StackPop());
        int x = AsInt(interp->StackPop());
        dim3 res(x, y, z);

        interp->StackPush(shared_ptr<S_Dim3>(new S_Dim3(res)));
    }
};


// ( dim3 -- coord )
class W_ToCoord : public Word
{
public:
    W_ToCoord(string name, string coord) : Word(name), coord(coord) {};

    virtual void Execute(Interpreter *interp) {
        dim3 d = AsDim3(interp->StackPop());

        int res = -1;
        if      (coord == "x")   res = d.x;
        else if (coord == "y")   res = d.y;
        else if (coord == "z")   res = d.z;
        else                     throw string("Unknown coord: ") + coord;

        interp->StackPush(shared_ptr<S_Int>(new S_Int(res)));
    }

protected:
    string coord;
};


// ( grid block -- )
class W_CheckIndex : public Word
{
public:
    W_CheckIndex(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        dim3 block = AsDim3(interp->StackPop());
        dim3 grid = AsDim3(interp->StackPop()); 

        checkIndex<<<grid, block>>>();
        hipDeviceReset();
    }
};


// ( type -- )
class W_Sizeof : public Word
{
public:
    W_Sizeof(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        string type = AsString(interp->StackPop());
        int result = 1;
        if      (type == "FLOAT")    result = sizeof(float);
        else if (type == "INT")      result = sizeof(int);
        interp->StackPush(shared_ptr<S_Int>(new S_Int(result)));
    }
};


// ( address offset num type -- )
class W_PrintMem : public Word
{
public:
    W_PrintMem(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        string type = AsString(interp->StackPop());
        int num = AsInt(interp->StackPop());
        int offset = AsInt(interp->StackPop());
        auto address = interp->StackPop();

        if (type == "FLOAT")    printMemAsFloats(AsFloatStar(address), offset, num);
        else                    printMemAsInts(AsIntStar(address), offset, num);
    }

protected:
    void printMemAsFloats(float* addr, int offset, int num) {
        for (int i=0; i < num ; i++) {
            printf("%-8.4f ", addr[offset+i]);
        }
    }

    void printMemAsInts(int* addr, int offset, int num) {
        for (int i=0; i < num ; i++) {
            printf("%-8d  ", addr[offset+i]);
        }
    }
};


void checkCudaCall(const hipError_t res, const char* file, int line) {
    if (res != hipSuccess) {
        stringstream builder;
        builder << hipGetErrorString(res) << " " << file << ":" << line;
        throw builder.str();
    }
}

// ( index -- )
class W_CudaSetDevice : public Word
{
public:
    W_CudaSetDevice(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int index = AsInt(interp->StackPop());
        auto res = hipSetDevice(index);
        checkCudaCall(res, __FILE__, __LINE__);
    }
};


// ( -- )
class W_CudaDeviceReset : public Word
{
public:
    W_CudaDeviceReset(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        auto res = hipDeviceReset();
        checkCudaCall(res, __FILE__, __LINE__);
    }
};


// ( num-bytes -- addr )
class W_CudaMalloc : public Word
{
public:
    W_CudaMalloc(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int num_bytes = AsInt(interp->StackPop());

        void *result;
        auto res = hipMalloc((void**)&result, num_bytes);
        checkCudaCall(res, __FILE__, __LINE__);
        interp->StackPush(S_Address::New(result));
    }
};


// ( num-bytes -- addr )
class W_CudaMallocManaged : public Word
{
public:
    W_CudaMallocManaged(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int num_bytes = AsInt(interp->StackPop());

        void *result;
        auto res = hipMallocManaged((void**)&result, num_bytes);
        checkCudaCall(res, __FILE__, __LINE__);
        interp->StackPush(S_Address::New(result));
    }
};


// ( addr -- )
class W_CudaFree : public Word
{
public:
    W_CudaFree(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        void* addr = AsVoidStar(interp->StackPop());
        auto res = hipFree(addr);
        checkCudaCall(res, __FILE__, __LINE__);
    }
};


// ( -- )
class W_CudaDeviceSynchronize : public Word
{
public:
    W_CudaDeviceSynchronize(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        auto res = hipDeviceSynchronize();
        checkCudaCall(res, __FILE__, __LINE__);
    }
};


// ( dst src num-bytes -- )
class W_CudaMemcpyHtD : public Word
{
public:
    W_CudaMemcpyHtD(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int num_bytes = AsInt(interp->StackPop());
        void* src = AsFloatStar(interp->StackPop());
        void* dst = AsFloatStar(interp->StackPop());

        auto res = hipMemcpy(dst, src, num_bytes, hipMemcpyHostToDevice);
        checkCudaCall(res, __FILE__, __LINE__);
    }
};


// ( dst src num-bytes -- )
class W_CudaMemcpyDtH : public Word
{
public:
    W_CudaMemcpyDtH(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int num_bytes = AsInt(interp->StackPop());
        void* src = AsFloatStar(interp->StackPop());
        void* dst = AsFloatStar(interp->StackPop());

        auto res = hipMemcpy(dst, src, num_bytes, hipMemcpyDeviceToHost);
        checkCudaCall(res, __FILE__, __LINE__);
    }
};


// ( devIndex -- hipDeviceProp_t )
class W_CudaGetDeviceProperties : public Word
{
public:
    W_CudaGetDeviceProperties(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        int devIndex = AsInt(interp->StackPop());

        hipDeviceProp_t deviceProp;
        auto res = hipGetDeviceProperties(&deviceProp, devIndex);
        checkCudaCall(res, __FILE__, __LINE__);
        interp->StackPush(S_CudaDeviceProp::New(deviceProp));
    }
};


// ( hipDeviceProp_t field -- value )
class W_DevProp : public Word
{
public:
    W_DevProp(string name) : Word(name) {};

    virtual void Execute(Interpreter *interp) {
        string field = AsString(interp->StackPop());
        shared_ptr<StackItem> item = interp->StackPop();

        if (auto devPropItem = dynamic_cast<S_CudaDeviceProp*>(item.get())) {
            const hipDeviceProp_t& deviceProp = devPropItem->deviceProp();
            if (field == "name") {
                interp->StackPush(S_String::New(string(deviceProp.name)));
            }
            else {
                throw string("Unknown dev prop field: ") + field;
            }
        }
        else {
            throw "Item was not a S_CudaDeviceProp";
        }
    }
};



// =============================================================================
// M_Cuda

M_Cuda::M_Cuda() : Module("cuda")
{
    AddWord(shared_ptr<Word>(new W_Dim3("DIM3")));
    AddWord(shared_ptr<Word>(new W_ToCoord(">x", "x")));
    AddWord(shared_ptr<Word>(new W_ToCoord(">y", "y")));
    AddWord(shared_ptr<Word>(new W_ToCoord(">z", "z")));
    AddWord(shared_ptr<Word>(new W_CheckIndex("GPU-CHECK-INDEX")));
    AddWord(shared_ptr<Word>(new W_Sizeof("SIZEOF")));
    AddWord(shared_ptr<Word>(new W_PrintMem("PRINT-MEM")));
    AddWord(shared_ptr<Word>(new W_CudaSetDevice("CUDA-SET-DEVICE")));
    AddWord(shared_ptr<Word>(new W_CudaDeviceReset("CUDA-DEVICE-RESET")));
    AddWord(shared_ptr<Word>(new W_CudaMalloc("CUDA-MALLOC")));
    AddWord(shared_ptr<Word>(new W_CudaMallocManaged("CUDA-MALLOC-MANAGED")));
    AddWord(shared_ptr<Word>(new W_CudaFree("CUDA-FREE")));
    AddWord(shared_ptr<Word>(new W_CudaDeviceSynchronize("CUDA-DEVICE-SYNCHRONIZE")));
    AddWord(shared_ptr<Word>(new W_CudaMemcpyHtD("CUDA-MEMCPY-HtD")));
    AddWord(shared_ptr<Word>(new W_CudaMemcpyDtH("CUDA-MEMCPY-DtH")));
    AddWord(shared_ptr<Word>(new W_CudaGetDeviceProperties("CUDA-GET-DEVICE-PROPERTIES")));
    AddWord(shared_ptr<Word>(new W_DevProp("DEV-PROP")));
}

string M_Cuda::ForthicCode() {
    string result(
    ": FLOAT   'FLOAT' ; "
    ": INT     'INT' ; "
    );
    return result;
}